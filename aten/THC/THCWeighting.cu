#include "hip/hip_runtime.h"
#include "THCWeighting.h"

#include "common.cuh"
#include "THCNumerics.cuh"

template<typename T>
__global__ void weightingForwardKernel(TensorInfo<T> self, TensorInfo<T> src, TensorInfo<T> weight,
                                       TensorInfo<T> basis, TensorInfo<int64_t> weightIndex,
                                       int n) {
  KERNEL_LOOP(i, n) {
    ptrdiff_t e = i / self.size[1], mOut = i % self.size[1], s, mIn;
    T v = ScalarConvert<int, T>::to(0), b, tmp;
    int64_t wi;
    for (s = 0; s < basis.size[1]; s++) {
      b = basis.data[e * basis.stride[0] + s * basis.stride[1]];
      wi = weightIndex.data[e * weightIndex.stride[0] + s * weightIndex.stride[1]];
      for (mIn = 0; mIn < src.size[1]; mIn++) {
        tmp = weight.data[wi * weight.stride[0] + mIn * weight.stride[1] + mOut * weight.stride[2]];
        tmp = THCNumerics<T>::mul(tmp, b);
        tmp = THCNumerics<T>::mul(tmp, src.data[e * src.stride[0] + mIn * src.stride[1]]);
        v = THCNumerics<T>::add(v, tmp);
      }
    }
    self.data[e * self.stride[0] + mOut * self.stride[1]] = v;
  }
}

#include "generic/THCWeighting.cu"
#include "THC/THCGenerateFloatTypes.h"
