#include "hip/hip_runtime.h"
#include "THCWeighting.h"

#include "common.cuh"
#include "THCNumerics.cuh"
#include "THCAtomics.cuh"

template<typename T>
__global__ void weightingForwardKernel(TensorInfo<T> self, TensorInfo<T> src, TensorInfo<T> weight,
                                       TensorInfo<T> basis, TensorInfo<int64_t> weightIndex,
                                       int n) {
  KERNEL_LOOP(i, n) {
    ptrdiff_t e = i / self.size[1], mOut = i % self.size[1], s, mIn;
    T v = ScalarConvert<int, T>::to(0), b, tmp;
    int64_t wi;
    for (s = 0; s < basis.size[1]; s++) {
      b = basis.data[e * basis.stride[0] + s * basis.stride[1]];
      wi = weightIndex.data[e * weightIndex.stride[0] + s * weightIndex.stride[1]];
      for (mIn = 0; mIn < src.size[1]; mIn++) {
        tmp = weight.data[wi * weight.stride[0] + mIn * weight.stride[1] + mOut * weight.stride[2]];
        tmp = THCNumerics<T>::mul(tmp, src.data[e * src.stride[0] + mIn * src.stride[1]]);
        tmp = THCNumerics<T>::mul(tmp, b);
        v = THCNumerics<T>::add(v, tmp);
      }
    }
    self.data[e * self.stride[0] + mOut * self.stride[1]] = v;
  }
}

template<typename T>
__global__ void weightingBackwardSrcKernel(TensorInfo<T> self, TensorInfo<T> gradOutput,
                                           TensorInfo<T> weight, TensorInfo<T> basis,
                                           TensorInfo<int64_t> weightIndex, int n) {
  KERNEL_LOOP(i, n) {
    ptrdiff_t e = i / gradOutput.size[1], mOut = i % gradOutput.size[1], s, mIn;
    T v, b, tmp;
    int64_t wi;
    T g = gradOutput.data[e * gradOutput.stride[0] + mOut * gradOutput.stride[1]];
    for (mIn = 0; mIn < self.size[1]; mIn++) {
      v = ScalarConvert<int, T>::to(0);
      for (s = 0; s < basis.size[1]; s++) {
        b = basis.data[e * basis.stride[0] + s * basis.stride[1]];
        wi = weightIndex.data[e * weightIndex.stride[0] + s * weightIndex.stride[1]];
        tmp = weight.data[wi * weight.stride[0] + mIn * weight.stride[1] + mOut * weight.stride[2]];
        tmp = THCNumerics<T>::mul(tmp, b);
        tmp = THCNumerics<T>::mul(tmp, g);
        v = THCNumerics<T>::add(v, tmp);
      }
      atomicAdd(&self.data[e * self.stride[0] + mIn * self.stride[1]], v);
    }
  }
}

template<typename T>
__global__ void weightingBackwardWeightKernel(TensorInfo<T> self, TensorInfo<T> gradOutput,
                                              TensorInfo<T> src, TensorInfo<T> basis,
                                              TensorInfo<int64_t> weightIndex, int n) {
  KERNEL_LOOP(i, n) {
    ptrdiff_t e = i / gradOutput.size[1], mOut = i % gradOutput.size[1], s, mIn;
    T b, v;
    int64_t wi;
    T g = gradOutput.data[e * gradOutput.stride[0] + mOut * gradOutput.stride[1]];
    for (s = 0; s < weightIndex.size[1]; s++) {
      b = basis.data[e * basis.stride[0] + s * basis.stride[1]];
      wi = weightIndex.data[e * weightIndex.stride[0] + s * weightIndex.stride[1]];
      for (mIn = 0; mIn < src.size[1]; mIn++) {
        v = src.data[e * src.stride[0] + mIn * src.stride[1]];
        v = THCNumerics<T>::mul(THCNumerics<T>::mul(v, b), g);
        atomicAdd(&self.data[wi * self.stride[0] + mIn * self.stride[1] + mOut * self.stride[2]], v);
      }
    }
  }
}

template<typename T>
__global__ void weightingBackwardBasisKernel(TensorInfo<T> self, TensorInfo<T> gradOutput,
                                             TensorInfo<T> src, TensorInfo<T> weight,
                                             TensorInfo<int64_t> weightIndex, int n) {
  KERNEL_LOOP(i, n) {
    ptrdiff_t e = i / gradOutput.size[1], mOut = i % gradOutput.size[1], s, mIn;
    T v, tmp;
    int64_t wi;
    T g = gradOutput.data[e * gradOutput.stride[0] + mOut * gradOutput.stride[1]];
    for (s = 0; s < weightIndex.size[1]; s++) {
      v = ScalarConvert<int, T>::to(0);
      wi = weightIndex.data[e * weightIndex.stride[0] + s * weightIndex.stride[1]];
      for (mIn = 0; mIn < src.size[1]; mIn++) {
        tmp = weight.data[wi * weight.stride[0] + mIn * weight.stride[1] + mOut * weight.stride[2]];
        tmp = THCNumerics<T>::mul(tmp, src.data[e * src.stride[0] + mIn * src.stride[1]]);
        tmp = THCNumerics<T>::mul(tmp, g);
        v = THCNumerics<T>::add(v, tmp);
      }
      atomicAdd(&self.data[e * self.stride[0] + s * self.stride[1]], v);
    }
  }
}

#include "generic/THCWeighting.cu"
#include "THC/THCGenerateFloatTypes.h"
