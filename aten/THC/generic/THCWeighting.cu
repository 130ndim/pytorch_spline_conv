
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCWeighting.cu"
#else

void THCTensor_(weightingForward)(THCState *state, THCTensor *self, THCTensor *src,
                                  THCTensor *weight, THCTensor *basis,
                                  THCudaLongTensor *weightIndex) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 5, self, src, weight, basis, weightIndex));

  TensorInfo<real> selfInfo = THCTensor_(getTensorInfo)(state, self);
  TensorInfo<real> srcInfo = THCTensor_(getTensorInfo)(state, src);
  TensorInfo<real> weightInfo = THCTensor_(getTensorInfo)(state, weight);
  TensorInfo<real> basisInfo = THCTensor_(getTensorInfo)(state, basis);
  TensorInfo<int64_t> weightIndexInfo = THCudaLongTensor_getTensorInfo(state, weightIndex);

  KERNEL_REAL_RUN(weightingForwardKernel, THCTensor_(nElement)(state, self), selfInfo, srcInfo,
                  weightInfo, basisInfo, weightIndexInfo);
}

void THCTensor_(weightingBackwardSrc)(THCState *state, THCTensor *self, THCTensor *gradOutput,
                                      THCTensor *weight, THCTensor *basis,
                                      THCudaLongTensor *weightIndex) {
}

void THCTensor_(weightingBackwardWeight)(THCState *state, THCTensor *self, THCTensor *gradOutput,
                                         THCTensor *src, THCTensor *basis,
                                         THCudaLongTensor *weightIndex) {
}

void THCTensor_(weightingBackwardBasis)(THCState *state, THCTensor *self, THCTensor *gradOutput,
                                        THCTensor *src, THCTensor *weight,
                                        THCudaLongTensor *weightIndex) {
}

#endif // THC_GENERIC_FILE

