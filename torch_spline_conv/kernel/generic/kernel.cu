
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

void spline_(linear_basis_forward)(THCState *state, THCTensor *basis, THCudaLongTensor *weight_index, THCTensor *pseudo, THCudaLongTensor *kernel_size, THCudaByteTensor *is_open_spline, int K) {
  SPLINE_BASIS_FORWARD(linearBasisForwardKernel, basis, weight_index, pseudo, kernel_size, is_open_spline, K)
}

void spline_(quadratic_basis_forward)(THCState *state, THCTensor *basis, THCudaLongTensor *weight_index, THCTensor *pseudo, THCudaLongTensor *kernel_size, THCudaByteTensor *is_open_spline, int K) {
  SPLINE_BASIS_FORWARD(quadraticBasisForwardKernel, basis, weight_index, pseudo, kernel_size, is_open_spline, K)
}

void spline_(cubic_basis_forward)(THCState *state, THCTensor *basis, THCudaLongTensor *weight_index, THCTensor *pseudo, THCudaLongTensor *kernel_size, THCudaByteTensor *is_open_spline, int K) {
  SPLINE_BASIS_FORWARD(cubicBasisForwardKernel, basis, weight_index, pseudo, kernel_size, is_open_spline, K)
}

#endif
