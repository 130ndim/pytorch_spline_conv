
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

void spline_(linear_basis_forward)(THCState *state, THCTensor *basis, THCudaLongTensor *weight_index, THCTensor *pseudo, THCudaLongTensor *kernel_size, THCudaByteTensor *is_open_spline, int K) {
  printf("linear");
}

void spline_(quadratic_basis_forward)(THCState *state, THCTensor *basis, THCudaLongTensor *weight_index, THCTensor *pseudo, THCudaLongTensor *kernel_size, THCudaByteTensor *is_open_spline, int K) {
  printf("quadratic");
}

void spline_(cubic_basis_forward)(THCState *state, THCTensor *basis, THCudaLongTensor *weight_index, THCTensor *pseudo, THCudaLongTensor *kernel_size, THCudaByteTensor *is_open_spline, int K) {
  printf("cubic");
}

#endif
