#include "hip/hip_runtime.h"
#include <THC.h>

#include "kernel.h"

#include "common.cuh"
#include "THCBasisForward.cuh"

#define spline_(NAME) TH_CONCAT_4(spline_, NAME, _kernel_, Real)
#define thc_(NAME) TH_CONCAT_4(thc_, NAME, _, Real)

#include "generic/common.cu"
#include "THCGenerateAllTypes.h"

template<typename Real>
__global__ void weightingForwardKernel(TensorInfo<Real> output, TensorInfo<Real> input, TensorInfo<Real> weight, TensorInfo<Real> basis, TensorInfo<int64_t> weightIndex, int n) {
  KERNEL_LOOP(i, n) {
    int64_t edgeOffset = i / output.size[1], inputOffset = edgeOffset * input.stride[0];
    int64_t s, S = basis.size[1], m_in, M_in = input.size[1], m_out = i % output.size[1], M_out = output.size[1], weightOffset;
    Real b, value = 0;
    for (s = 0; s < S; s++) {
      b = basis.data[edgeOffset + s];
      weightOffset = weightIndex.data[edgeOffset * S + s] * M_in * M_out + m_out;
      for (m_in = 0; m_in < M_in; m_in++) {
        value += b * weight.data[weightOffset + m_in * M_out] * input.data[inputOffset + m_in * input.stride[1]];
      }
    }
    output.data[i] = value;
  }
}

#include "generic/kernel.cu"
#include "THCGenerateFloatType.h"
#include "generic/kernel.cu"
#include "THCGenerateDoubleType.h"
